#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

int main() {
    const unsigned int N = 1048576*16;
    const unsigned int bytes = N * sizeof(int);
    int *h_a;
    int *d_a;
    hipMalloc((int**)&d_a, bytes);
	hipError_t status = hipHostMalloc((void**)&h_a, bytes, hipHostMallocDefault);
	if (status != hipSuccess)
		printf("Error allocating pinned host memory\n");
 
    memset(h_a, 0, bytes);
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);

    return 0;
}
