#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y, int chunk_thread) {
	int chunk_block = blockDim.x*chunk_thread;
	int from_n = chunk_block*blockIdx.x + chunk_thread*threadIdx.x;
	int to_n = min(n, from_n+chunk_thread);
	for (int i = from_n; i < to_n; i++)
		y[i] = x[i] + y[i];
}

int main(void) {
	int N = (1<<20);
	float *x, *y;

	// Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));

	// initialize x and y arrays on the host
	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	int blockSize = 256;
	//int numBlocks = (N + blockSize - 1) / blockSize;
	int numBlocks = 8;
	// Run kernel on 1M elements on the GPU
	const int chunk_thread = (N+numBlocks*blockSize-1)/numBlocks/blockSize;

	add<<<numBlocks, blockSize>>>(N, x, y, chunk_thread);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	// Check for errors (all values should be 3.0f)
	float maxError = 0.0f;
	for (int i = 0; i < N; i++)
		maxError = fmax(maxError, fabs(y[i]-3.0f));
	std::cout << "Max error: " << maxError << std::endl;

	// Free memory
	hipFree(x);
	hipFree(y);
  
	return 0;
}
