#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "transpose.hh"
#include "detect.hh"
#include "die.h"
#include <stdio.h>
#include <hip/device_functions.h>
#include <algorithm>

constexpr int THREADS = 128;

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
static hipError_t checkCuda(hipError_t result) {
	if (result != hipSuccess)
		die("CUDA Runtime Error: %s\n", hipGetErrorString(result));
	return result;
}

/**
 * detect a sequence of ascending chars
 */
__global__ void detectSeq(Link *odata, const char *idata, unsigned *d_nlink, unsigned dim) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int ofs = col;
	int len = 0;
	for (unsigned row=0; row<dim; row++,ofs+=dim) {
		if (idata[ofs] == ' ')
			len++;
		odata[ofs].prev = 1;
	}
	d_nlink[col] = col;
}

__global__ void reduceMax(const unsigned *d_in, unsigned *d_out, unsigned dim) {
	__shared__ int rmx[THREADS/32]; // current warp size = 32
 	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int mx = d_in[col];
	for (int mask = warpSize/2; mask > 0; mask /= 2) 
		mx = max(mx, __shfl_xor(mx, mask));
	int w = threadIdx.x / warpSize;
	int lane = threadIdx.x % warpSize;
	if (lane == 0)
		rmx[w] = mx;
	__syncthreads();
	mx = (threadIdx.x < THREADS/warpSize) ? rmx[lane] : 0;
	for (int mask = warpSize/2; mask > 0; mask /= 2) 
		mx = max(mx, __shfl_xor(mx, mask));	
	if (threadIdx.x == 0)
		d_out[blockIdx.x] = mx;
}

Match::Detect::Detect(const char *_d_ibuf):d_ibuf(_d_ibuf),d_obuf(nullptr),obuf(nullptr),d_nlink(nullptr),d_nlink_block(nullptr),width(0) {
}

int Match::Detect::init(Link *ob, int w) {
	obuf= ob;
	width = w;
	if (w%THREADS != 0)
		die("width must be divisible by %d", THREADS);
	int rc;
	if ((rc=hipMalloc(&d_obuf, w*w*sizeof(Link)))!=0)
		return rc;
	if ((rc=hipMalloc(&d_nlink, w*sizeof(d_nlink[0])))!=0)
		return rc;
	if ((rc=hipMalloc(&d_nlink_block, w/THREADS*sizeof(d_nlink[0])))!=0)
		return rc;
	return 0;
}

Match::Detect::~Detect() {
	if (d_obuf)
		checkCuda(hipFree((void*)d_obuf));
	if (d_nlink)
		checkCuda(hipFree((void*)d_nlink));
	if (d_nlink_block)
		checkCuda(hipFree((void*)d_nlink_block));
}

unsigned Match::Detect::run() {
	int blocks = width/THREADS;
	detectSeq<<<blocks,THREADS>>>((Link*)d_obuf, (const char*)d_ibuf, d_nlink, width);
	reduceMax<<<blocks,THREADS>>>(d_nlink, d_nlink_block, width);
	reduceMax<<<1,blocks>>>(d_nlink_block, d_nlink_block, width);
	checkCuda(hipMemcpy(obuf, d_obuf, width*width*sizeof(obuf[0]), hipMemcpyDeviceToHost));
	checkCuda(hipMemcpy(&nlinkmax, d_nlink_block, sizeof(unsigned), hipMemcpyDeviceToHost));
	return nlinkmax;
}
