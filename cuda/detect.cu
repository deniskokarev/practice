#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "transpose.hh"
#include "detect.hh"
#include "die.h"
#include <stdio.h>

constexpr int THREADS = 128;

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
static hipError_t checkCuda(hipError_t result) {
	if (result != hipSuccess)
		die("CUDA Runtime Error: %s\n", hipGetErrorString(result));
	return result;
}

/**
 * detect a sequence of ascending chars
 */
__global__ void detectSeq(Link *odata, const char *idata, unsigned *d_nlink, unsigned dim) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int ofs = col;
	int len = 0;
	for (unsigned row=0; row<dim; row++,ofs+=dim) {
		if (idata[ofs] == ' ')
			len++;
		odata[ofs].prev = 1;
	}
}

__global__ void max(const unsigned *d_in, unsigned *d_out, unsigned dim) {
	unsigned mx = 0;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < dim; i += blockDim.x * gridDim.x)
		mx = max(mx, d_in[i]);
	d_out[blockIdx.x] = mx;
}

Match::Detect::Detect(const char *_d_ibuf):d_ibuf(_d_ibuf),d_obuf(nullptr),obuf(nullptr),d_nlink(nullptr),d_nlink_block(nullptr),width(0) {
}

int Match::Detect::init(Link *ob, int w) {
	obuf= ob;
	width = w;
	if (w%THREADS != 0)
		die("width must be divisible by %d", THREADS);
	int rc;
	if ((rc=hipMalloc(&d_obuf, w*w*sizeof(Link)))!=0)
		return rc;
	if ((rc=hipMalloc(&d_nlink, w*sizeof(d_nlink[0])))!=0)
		return rc;
	if ((rc=hipMalloc(&d_nlink_block, w/THREADS*sizeof(d_nlink[0])))!=0)
		return rc;
	return 0;
}

Match::Detect::~Detect() {
	if (d_obuf)
		checkCuda(hipFree((void*)d_obuf));
	if (d_nlink)
		checkCuda(hipFree((void*)d_nlink));
	if (d_nlink_block)
		checkCuda(hipFree((void*)d_nlink_block));
}

unsigned Match::Detect::run() {
	int blocks = width/THREADS;
	detectSeq<<<blocks,THREADS>>>((Link*)d_obuf, (const char*)d_ibuf, d_nlink, width);
	max<<<blocks,THREADS>>>(d_nlink, d_nlink_block, width);
	max<<<1,blocks>>>(d_nlink_block, d_nlink_block, width);
	checkCuda(hipMemcpy(obuf, d_obuf, width*width*sizeof(obuf[0]), hipMemcpyDeviceToHost));
	checkCuda(hipMemcpy(&nlinkmax, d_nlink_block, sizeof(unsigned), hipMemcpyDeviceToHost));
	return nlinkmax;
}
