#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "transpose.hh"
#include "detect.hh"
#include "die.h"
#include <stdio.h>
#include <hip/device_functions.h>
#include <algorithm>

constexpr int THREADS = 256;

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
static hipError_t checkCuda(hipError_t result) {
	if (result != hipSuccess)
		die("CUDA Runtime Error: %s\n", hipGetErrorString(result));
	return result;
}

/**
 * detect a sequence of ascending chars
 */
__global__ void detectSeq(Link *odata, char *idata, unsigned *d_nlink, uint16_t dim) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int ofs = col;
	uint16_t nlink = 0;
	char ch = 0;
	char match = 0;
	Link l;
	l.len = 0;
	l.pos = 0;
	l.next = 0;
	for (uint16_t row=0; row<dim; row++,ofs+=dim) {
		if (idata[ofs] == ch) {
			l.len++;
			match = 1;
		} else {
			if (match) {
				l.len++;
				odata[ofs] = l;
				nlink++;
				l.next = row;
			}
			l.len = 0;
			match = 0;
		}
		ch = idata[ofs]+1;
	}
	d_nlink[col] = nlink;
	if (l.len > 0) {
		l.next = dim;
		l.len++;
	}
	odata[ofs] = l;
	ofs = col+dim*dim;
	int oofs = col+dim*(dim-1);
	uint16_t next = 0;
	uint16_t left = nlink;
	for (uint16_t row=dim-1; row>0; row--,ofs-=dim) {
		if (row == l.next) {
			l = odata[ofs];
			odata[oofs].len = l.len;
			odata[oofs].pos = row-l.len;
			odata[oofs].next = next;
			oofs--;
			left--;
			next = row;
		}
	}
	l.len = 0;
	l.pos = 0;
	l.next = next;
	ofs = dim*dim+col;
	odata[ofs] = l;
}

__global__ void recalcLinksSeq(Link *odata, uint16_t dim, uint16_t nlink) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int zrow = dim-nlink-1;
	int erow = dim;
	Link l = odata[erow*dim+col];
	odata[zrow*dim+col] = l;
	int ofs = zrow*dim+col;
	for (int row=zrow; row<dim; row++,ofs+=dim)
		odata[ofs].next -= zrow;
}

__global__ void reduceMax(const unsigned *d_in, unsigned *d_out, unsigned dim) {
	__shared__ int rmx[THREADS/32]; // current warp size = 32
 	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int mx = d_in[col];
	for (int mask = warpSize/2; mask > 0; mask /= 2) 
		mx = max(mx, __shfl_xor(mx, mask));
	int w = threadIdx.x / warpSize;
	int lane = threadIdx.x % warpSize;
	if (lane == 0)
		rmx[w] = mx;
	__syncthreads();
	// since warpSize*warpSize >= THREADS, run second stage right here
	mx = (lane < THREADS / warpSize) ? rmx[lane] : 0;
	for (int mask = warpSize/2; mask > 0; mask /= 2) 
		mx = max(mx, __shfl_xor(mx, mask));	
	if (threadIdx.x == 0)
		d_out[blockIdx.x] = mx;
}

Match::Detect::Detect(const char *_d_ibuf, Link *ob, int w):d_ibuf(_d_ibuf),d_obuf(nullptr),obuf(ob),d_nlink(nullptr),d_nlink_block(nullptr),width(w),err(nullptr) {
	if (w%THREADS == 0) {
		hipError_t rc;
		if ((rc=hipMalloc(&d_obuf, (w+1)*w*sizeof(Link))) != 0)
			err = hipGetErrorString(rc);
		if ((rc=hipMalloc(&d_nlink, w*sizeof(int))) != 0)
			err = hipGetErrorString(rc);
		if ((rc=hipMalloc(&d_nlink_block, w/THREADS*sizeof(int))) != 0)
			err = hipGetErrorString(rc);
	} else {
		snprintf(errbuf, sizeof(errbuf), "width must be divisible by %d", THREADS);
		err = errbuf;
	}
}

Match::Detect::operator bool() const {
	return err == nullptr;
}

Match::Detect::~Detect() {
	if (d_nlink_block)
		checkCuda(hipFree((void*)d_nlink_block));
	if (d_nlink)
		checkCuda(hipFree((void*)d_nlink));
	if (d_obuf)
		checkCuda(hipFree((void*)d_obuf));
}

unsigned Match::Detect::run() {
	int blocks = width/THREADS;
	detectSeq<<<blocks,THREADS>>>((Link*)d_obuf, (char*)d_ibuf, d_nlink, width);
	checkCuda(hipGetLastError());
	reduceMax<<<blocks,THREADS>>>(d_nlink, d_nlink_block, width);
	checkCuda(hipGetLastError());
	reduceMax<<<1,blocks>>>(d_nlink_block, d_nlink_block, width);
	checkCuda(hipGetLastError());
	checkCuda(hipMemcpy(&nlinkmax, d_nlink_block, sizeof(unsigned), hipMemcpyDeviceToHost));
	recalcLinksSeq<<<blocks,THREADS>>>((Link*)d_obuf, width, nlinkmax);
	checkCuda(hipMemcpy(obuf, d_obuf+(width-nlinkmax)*width, nlinkmax*width*sizeof(obuf[0]), hipMemcpyDeviceToHost));
	return nlinkmax;
}
