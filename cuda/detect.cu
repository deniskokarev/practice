#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "transpose.hh"
#include "detect.hh"
#include "die.h"
#include <stdio.h>
#include <hip/device_functions.h>
#include <algorithm>

constexpr int THREADS = 1024;

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
static hipError_t checkCuda(hipError_t result) {
	if (result != hipSuccess)
		die("CUDA Runtime Error: %s\n", hipGetErrorString(result));
	return result;
}

__device__ void compactLinks(Link *odata, const uint16_t dim);

/**
 * detect a sequence of ascending chars
 */
__global__ void detectSeq(Link *odata, const char *idata, unsigned *d_nlink, uint16_t dim) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int ofs = col;
	uint16_t nlink = 0;
	char expch = 0;
	Link l;
	l.len = 0;
	l.c.prev = 0;
	uint16_t row;
	for (row=0; row<dim; row++,ofs+=dim) {
		if (idata[ofs] == expch) {
			l.len++;
		} else {
			if (l.len>0) {
				l.len++;
				odata[ofs] = l;
				nlink++;
				l.c.prev = row;
			}
			l.len = 0;
		}
		expch = idata[ofs]+1;
	}
	if (l.len>0) {
		l.len++;
		odata[ofs] = l;
		nlink++;
		l.c.prev = row;
	}
	ofs += dim;
	odata[ofs] = l;
	d_nlink[col] = nlink;
	compactLinks(odata, dim);
}

/**
 * "compact" linked-lists into sequential entries
 */
__device__ void compactLinks(Link *odata, const uint16_t dim) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int ofs = (dim+1)*dim+col;
	int oofs = dim*dim+col;
	Link l = odata[ofs];
	ofs = oofs;
	int nlink = 0;
	for (uint16_t row=dim; row>0; row--,ofs-=dim) {
		if (row == l.c.prev) {
			l = odata[ofs];
			odata[oofs] = {l.len, {uint16_t(row-l.len)}};
			oofs -= dim;
			nlink++;
		}
	}
	uint16_t startrow = dim-nlink;
	odata[col].len = nlink;
	oofs = dim+col;
	ofs = dim+col;
	for (uint16_t row=1; row<=dim; row++,ofs+=dim) {
		if (row > startrow) {
			odata[oofs] = odata[ofs];
			oofs += dim;
		}
	}
}

__global__ void reduceMax(const unsigned *d_in, unsigned *d_out, unsigned dim) {
	__shared__ int rmx[THREADS/32]; // current warp size = 32
 	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int mx = d_in[col];
	for (int mask = warpSize/2; mask > 0; mask /= 2) 
		mx = max(mx, __shfl_xor(mx, mask));
	int w = threadIdx.x / warpSize;
	int lane = threadIdx.x % warpSize;
	if (lane == 0)
		rmx[w] = mx;
	__syncthreads();
	// since warpSize*warpSize >= THREADS, run second stage right here
	mx = (lane < THREADS / warpSize) ? rmx[lane] : 0;
	for (int mask = warpSize/2; mask > 0; mask /= 2) 
		mx = max(mx, __shfl_xor(mx, mask));	
	if (threadIdx.x == 0)
		d_out[blockIdx.x] = mx;
}

Match::Detect::Detect(const char *_d_ibuf, Link *ob, int w):d_ibuf(_d_ibuf),d_obuf(nullptr),obuf(ob),d_nlink(nullptr),d_nlink_block(nullptr),width(w),err(nullptr) {
	if (w%THREADS == 0) {
		hipError_t rc;
		if ((rc=hipMalloc(&d_obuf, (w+2)*w*sizeof(Link))) != 0)
			err = hipGetErrorString(rc);
		if ((rc=hipMalloc(&d_nlink, w*sizeof(int))) != 0)
			err = hipGetErrorString(rc);
		if ((rc=hipMalloc(&d_nlink_block, w/THREADS*sizeof(int))) != 0)
			err = hipGetErrorString(rc);
	} else {
		snprintf(errbuf, sizeof(errbuf), "width must be divisible by %d", THREADS);
		err = errbuf;
	}
}

Match::Detect::operator bool() const {
	return err == nullptr;
}

Match::Detect::~Detect() {
	if (d_nlink_block)
		checkCuda(hipFree((void*)d_nlink_block));
	if (d_nlink)
		checkCuda(hipFree((void*)d_nlink));
	if (d_obuf)
		checkCuda(hipFree((void*)d_obuf));
}

unsigned Match::Detect::run() {
	int blocks = width/THREADS;
	detectSeq<<<blocks,THREADS>>>(d_obuf, d_ibuf, d_nlink, width);
	checkCuda(hipGetLastError());
	reduceMax<<<blocks,THREADS>>>(d_nlink, d_nlink_block, width);
	checkCuda(hipGetLastError());
	reduceMax<<<1,blocks>>>(d_nlink_block, d_nlink_block, width);
	checkCuda(hipGetLastError());
	checkCuda(hipMemcpy(&nlinkmax, d_nlink_block, sizeof(unsigned), hipMemcpyDeviceToHost));
	checkCuda(hipMemcpy(obuf, d_obuf, (nlinkmax+1)*width*sizeof(obuf[0]), hipMemcpyDeviceToHost));
	return nlinkmax;
}
