#include "hip/hip_runtime.h"
/**
 * a-la fgrep utility only in parallel using CUDA
 * grep input lines for precompiled patterns
 * @author Denis Kokarev
 */
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <memory>
#include <stdarg.h>
#include <getopt.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include "transpose.cuh"
#include "die.h"
#include "par.hh"
#include "act.h"

constexpr int THREADS = 256;
constexpr int STRSZ = 1<<14; // must be under int16
constexpr int STREAMS = STRSZ;

struct MATCH {
	int16_t pos;
	uint16_t sz;
};

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
#define checkCuda(e) {	if (e!=hipSuccess) { die("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e)); } }

// owns input the data buffers for the pipeline
class ReadStage: public PipeHeadExec {
public:
	struct TRESULT {
		char *buf;
		int sz;
	};
private:
	static constexpr int stages = 4; // need to drag the input over 4 pipe segments
	FILE *fin;
	TRESULT res[stages];
	char *pinned_buf;
private:
	virtual void *next() override {
		if (!feof(fin)) {
			if (batch%stages == 0) // wrap the remaining line around
				memcpy(res[0].buf-STRSZ, res[stages-1].buf+STREAMS*STRSZ-STRSZ, STRSZ);
			TRESULT &r = res[batch%stages];
			r.sz = fread(r.buf, 1, STREAMS*STRSZ, fin);
			if (r.sz < 0)
				die("Read error");
			return &r;
		} else {
			return nullptr;
		}
	}
public:
	ReadStage(FILE *fin):PipeHeadExec(), fin(fin) {
		checkCuda(hipHostMalloc(&pinned_buf, STRSZ*STREAMS*stages+STRSZ));
		res[0] = {pinned_buf+STRSZ, 0};
		for (int i=1; i<stages; i++)
			res[i] = {res[i-1].buf+STRSZ*STREAMS, 0};
	}
	~ReadStage() {
		checkCuda(hipHostFree(pinned_buf));
	}
};

class CudaH2DStage: public PipeStageExec {
public:
	struct TRESULT {
		ReadStage::TRESULT in;
		char *d_ibuf; // device original input
		hipStream_t stream;
	};
private:
	static constexpr int stages = 2;
	TRESULT res[stages];
	virtual void *next(void *arg) override {
		TRESULT &r = res[batch%stages];
		r.in = *(ReadStage::TRESULT*)arg;
		checkCuda(hipMemcpyAsync(r.d_ibuf, r.in.buf, r.in.sz, hipMemcpyHostToDevice, r.stream));
		checkCuda(hipStreamSynchronize(r.stream));
		return &r;
	}
public:
	CudaH2DStage(PipeHeadExec &parent):PipeStageExec(parent) {
		for (int i=0; i<stages; i++) {
			res[i].in = ReadStage::TRESULT {nullptr, 0};
			checkCuda(hipMalloc(&res[i].d_ibuf, STREAMS*STRSZ*sizeof(*res[i].d_ibuf)));
			checkCuda(hipStreamCreate(&res[i].stream));
		}
	}
	~CudaH2DStage() {
		for (int i=0; i<stages; i++) {
			res[i].in = ReadStage::TRESULT {nullptr, 0};
			checkCuda(hipFree(res[i].d_ibuf));
			checkCuda(hipStreamDestroy(res[i].stream));
		}
	}
};

struct FGREP_STATE {
	unsigned node;
	int16_t lbeg;	// position where last line started
	uint16_t match;
};

/**
 * consume next char
 * @returns next automata node, where 1 means at root
 * only when at root, it is safe to refresh/reattach the trie
 */
__device__ inline unsigned cuda_act_next_char(const ACT_NODE *act, unsigned node, char ch) {
	unsigned bmask = ch;
	for (int i=0; i<8/ACT_PAGE_P2; i++,bmask >>= ACT_PAGE_P2)
		node = act[node].next[bmask & ((1<<ACT_PAGE_P2)-1)];
	return node;
}

/**
 * iterate the sequence of all matches triggered by last char
 * NB: result_node will be spoiled
 * @return 1 when match was found, *val will be populated with value
 * @return 0 when no more matches
 */
__device__ inline int cuda_act_next_match(const ACT_NODE *act, unsigned *result_node, int *val) {
	if (!act[*result_node].end)
		*result_node = act[*result_node].sufref;
	if (*result_node != ACT_ROOT) {
		*val = act[*result_node].val;
		*result_node = act[*result_node].sufref;
		return 1;
	} else {
		return 0;
	}
}

struct CHAR_BUF {
	const char * const ibuf;
	const int ibufsz;
	int pos;
	int row;
	int col;
	const int stride;
	const char *s;
};

// *s++ in our transposed buffer
__device__ inline short ch_next(CHAR_BUF &ch) {
	if (ch.pos < ch.ibufsz) {
		short c = *ch.s;
		ch.pos++;
		ch.row++;
		ch.s += ch.stride;
		if (ch.row == STRSZ) {
			ch.col++;
			ch.row = 0;
			ch.s = ch.ibuf+ch.col;
		}
		return c;
	} else {
		return -1;
	}
}

__device__ inline short ch_seek_nl(CHAR_BUF &ch, unsigned limit) {
	short c = -1;
	while (limit && (c=ch_next(ch))>=0 && (c != '\n'))
		limit--;
	return c;
}

__global__ void cuda_fgrep(MATCH *match, const char *ibuf, int ibufsz, unsigned *nmatch, const ACT_NODE *act, FGREP_STATE *states) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = gridDim.x * blockDim.x; // STREAMS
	MATCH *m = match+col;
	unsigned nm = 0;		// number of matches > STRSZ works as error indicator
	FGREP_STATE state;
	CHAR_BUF ch { ibuf, ibufsz-col*STRSZ, 0, 0, col, stride, ibuf+col };
	//__syncthreads(); // redundant, as the first thread will always run in an earlier block
	short c;
	if (col == 0) {
		state = states[STREAMS-1];
		c = ch_next(ch);
	} else {
		if ((c=ch_seek_nl(ch, STRSZ)) != '\n') {
			state = FGREP_STATE {ACT_ROOT, 0, 0};
			c = -1;
		}
	}
	while (c >= 0) {
		if (c == '\n') {
			if (state.match) {
				*m = MATCH {state.lbeg, uint16_t(ch.pos-state.lbeg-1)};
				nm++;
				m += stride;
			}
			state = FGREP_STATE {ACT_ROOT, int16_t(ch.pos), 0};
			if (ch.row > 0 && ch.col > col)
				break;
		}
		if (!state.match) {
			state.node = cuda_act_next_char(act, state.node, c);
			unsigned result_node = state.node;
			int unused;
			state.match = cuda_act_next_match(act, &result_node, &unused);
		}
		c = ch_next(ch);
	}
	state.lbeg -= STRSZ;
	states[col] = state;
	nmatch[col] = nm;
}

class CudaFgrep {
	static constexpr int MATCH_RATIO = 10; // lowest ave line len to save mem
	char *d_tibuf; // transposed input
	MATCH *d_tobuf; // transposed output
	MATCH *d_obuf; // regular output
	unsigned *d_nmatch;
	FGREP_STATE *d_state;
	ACT_NODE *d_act;
public:
	CudaFgrep(const ACT *act) {
		checkCuda(hipMalloc(&d_tibuf, STREAMS*STRSZ*sizeof(*d_tibuf)));
		checkCuda(hipMalloc(&d_tobuf, STREAMS*(STRSZ/MATCH_RATIO)*sizeof(*d_tobuf)));
		checkCuda(hipMalloc(&d_obuf, STREAMS*(STRSZ/MATCH_RATIO)*sizeof(*d_tobuf)));
		checkCuda(hipMalloc(&d_nmatch, STREAMS*sizeof(*d_nmatch)));
		checkCuda(hipMalloc(&d_state, STREAMS*sizeof(*d_state)));
		FGREP_STATE first_state {ACT_ROOT, 0, 0};
		checkCuda(hipMemcpy(&d_state[STREAMS-1], &first_state, sizeof(FGREP_STATE), hipMemcpyHostToDevice));
		checkCuda(hipMalloc(&d_act, act->sz));
		checkCuda(hipMemcpy(d_act, act->nodes, act->sz, hipMemcpyHostToDevice));
	}
	~CudaFgrep() {
		checkCuda(hipFree(d_tibuf));
		checkCuda(hipFree(d_tobuf));
		checkCuda(hipFree(d_obuf));
		checkCuda(hipFree(d_nmatch));
		checkCuda(hipFree(d_state));
		checkCuda(hipFree(d_act));
	}
	void operator()(hipStream_t stream, const char *d_ibuf, int ibuf_sz, MATCH *obuf, unsigned *nmatch, unsigned &rowsz) {
		dim3 dimGrid(STRSZ/TRANSPOSE_TILE_DIM, STREAMS/TRANSPOSE_TILE_DIM, 1);
		dim3 dimBlock(TRANSPOSE_TILE_DIM, TRANSPOSE_BLOCK_ROWS, 1);
		transposeNoBankConflicts<<<dimGrid, dimBlock, 0, stream>>>(d_tibuf, d_ibuf);
		checkCuda(hipGetLastError());
		cuda_fgrep<<<STREAMS/THREADS,THREADS,0,stream>>>(d_tobuf, d_tibuf, ibuf_sz, d_nmatch, d_act, d_state);
		checkCuda(hipGetLastError());
		checkCuda(hipMemcpyAsync(nmatch, d_nmatch, sizeof(*nmatch)*STREAMS, hipMemcpyDeviceToHost, stream));
		checkCuda(hipStreamSynchronize(stream));
		unsigned nmx = rowsz = *std::max_element(nmatch, nmatch+STREAMS);
		if (nmx > 0) {
			if (nmx > STRSZ)
				die("Lines cannot be longer than %d", int(STRSZ));
			rowsz = (nmx+TRANSPOSE_TILE_DIM-1)/TRANSPOSE_TILE_DIM*TRANSPOSE_TILE_DIM;
			dim3 dimGrid(STREAMS/TRANSPOSE_TILE_DIM, rowsz/TRANSPOSE_TILE_DIM, 1);
			dim3 dimBlock(TRANSPOSE_TILE_DIM, TRANSPOSE_BLOCK_ROWS, 1);
			transposeNoBankConflicts<<<dimGrid, dimBlock, 0, stream>>>(d_obuf, d_tobuf);
			checkCuda(hipGetLastError());
			checkCuda(hipMemcpyAsync(obuf, d_obuf, rowsz*STREAMS*sizeof(obuf[0]), hipMemcpyDeviceToHost, stream));
			checkCuda(hipStreamSynchronize(stream));
		}
	}
};

class FgrepStage: public PipeStageExec {
public:
	struct TRESULT {
		CudaH2DStage::TRESULT in;
		unsigned match_row_sz;
		MATCH *match;
		unsigned *nmatch;
	};
private:
	static constexpr int stages = 2;
	TRESULT res[stages];
	CudaFgrep cfgrep;
	virtual void *next(void *arg) override {
		TRESULT &r = res[batch%stages];
		r.in = *(CudaH2DStage::TRESULT*)arg;
		cfgrep(r.in.stream, r.in.d_ibuf, r.in.in.sz, r.match, r.nmatch, r.match_row_sz);
		return &r;
	}
public:
	FgrepStage(PipeHeadExec &parent, ACT *act):PipeStageExec(parent),cfgrep(act) {
		for (int i=0; i<stages; i++) {
			checkCuda(hipHostMalloc(&res[i].match, sizeof(MATCH)*STRSZ*STREAMS));
			checkCuda(hipHostMalloc(&res[i].nmatch, sizeof(unsigned)*STREAMS));
		}
	}
	~FgrepStage() {
		for (int i=0; i<stages; i++) {
			checkCuda(hipHostFree(res[i].match));
			checkCuda(hipHostFree(res[i].nmatch));
		}
	}
};

void prn(FILE *fout, const char *ibuf, const MATCH *match, const unsigned *nmatch, const unsigned match_row_sz) {
	for (int stream=0; stream<STREAMS; stream++) {
		unsigned sz = nmatch[stream];
		const MATCH *mm = match+match_row_sz*stream;
		const char *s = ibuf+STRSZ*stream;
		for (unsigned i=0; i<sz; i++) {
			if ((fwrite(s+mm[i].pos, 1, mm[i].sz, fout)) != (int)mm[i].sz)
				die("Write error");
			if (fputc('\n', fout) != '\n')
				die("Write error");
		}
	}
}

static void usage(char *cmd) {
	printf("Match the input strings with actcomp precompiled automata,\n");
	printf("works similar to fgrep\n");
	printf("@author Denis Kokarev\n");
	printf("Usage:\n");
	printf("\t%s patterns.bin <input.txt >filtered.txt\n", cmd);
	printf("patterns.bin - precompiled patterns.txt file, see `actcomp -h`\n");
}

int main(int argc, char **argv) {
	int c;
	opterr = 0;
	while ((c = getopt(argc, argv, "h")) != -1) {
		switch (c) {
		case 'h':
			usage(argv[0]);
			return(0);
		default:
			die("unknown cmd line argument");
		}
	}
	if (argc-optind < 1) {
		usage(argv[0]);
		die("run as `%s patterns.bin <file`", argv[0]);
	}
	ACT act;
	int rc = act_attach_mmap(&act, argv[optind]);
	if (rc != 0)
		die("couldn't use specified patterns file %s, act_attach_mmap() error code %d", argv[optind], rc);
	{
		ReadStage read(stdin);
		CudaH2DStage h2d(read);
		FgrepStage fgrep(h2d, &act);
		for (auto it:PipeOutput(fgrep)) {
			FgrepStage::TRESULT *r = (FgrepStage::TRESULT*)it;
			prn(stdout, r->in.in.buf, r->match, r->nmatch, r->match_row_sz);
		}
	}
	rc = act_detach_mmap(&act);
	if (rc != 0)
		die("act_detach_mmap() error code %d", rc);
}
