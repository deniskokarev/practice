#include "hip/hip_runtime.h"
/**
 * similar to uuidgrep, only
 * extracting UUIDs from stream of ASCII strings
 * use STREAMS parallel threads for speed
 * CUDA version of the tool
 * @author Denis Kokarev
 */
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <memory>
#include <algorithm>
#include <hip/hip_runtime.h>
#include "transpose.cuh"
#include "uuidmatch.h"
#include "die.h"
#include "par.hh"

constexpr int THREADS = 256;
constexpr int STRSZ = 1<<14; // must be under int16
constexpr int STREAMS = STRSZ;

struct MATCH {
	int16_t pos;
	uint16_t sz;
};

// owns input the data buffers for the pipeline
class ReadStage: public PipeHeadExec {
public:
	struct TRESULT {
		char *buf;
		int sz;
	};
private:
	static constexpr int stages = 3; // need to drag the input over 3 pipe segments
	FILE *fin;
	TRESULT res[stages];
	std::unique_ptr<char[]> up_buf;
private:
	virtual void *next() override {
		if (!feof(fin)) {
			if (batch%stages == 0) // wrap the remaining line around
				memcpy(res[0].buf-UMPATLEN, res[stages-1].buf+STREAMS*STRSZ-UMPATLEN, UMPATLEN);
			TRESULT &r = res[batch%stages];
			r.sz = fread(r.buf, 1, STREAMS*STRSZ, fin);
			if (r.sz < 0)
				die("Read error");
			return &r;
		} else {
			return nullptr;
		}
	}
public:
	ReadStage(FILE *fin):PipeHeadExec(),
						 fin(fin),
						 up_buf(new char[STRSZ*STREAMS*stages+STRSZ]) {
		res[0] = {up_buf.get()+STRSZ, 0};
		for (int i=1; i<stages; i++)
			res[i] = {res[i-1].buf+STRSZ*STREAMS, 0};
	}
};

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
static hipError_t checkCuda(hipError_t result) {
	if (result != hipSuccess)
		die("CUDA Runtime Error: %s\n", hipGetErrorString(result));
	return result;
}

inline __device__ int cuda_ch_category(unsigned char c) {
	if ((c>='0' && c<='9') || (c>='a' && c<='f') || (c>='A' && c<='F'))
		return P_HEX;
	else if (c == '-')
		return P_DASH;
	else
		return P_NONE;
}

__constant__ UMSTATE cuda_uuid_pattern[UMPATLEN+1][P_SZ];

// return true when ch was the last char in uuid pattern
inline __device__ int cuda_um_match(UMSTATE *state, char ch) {
	int cat = cuda_ch_category(ch);
	*state = cuda_uuid_pattern[*state][cat];
	if (*state == UMPATLEN) {
		*state = cuda_uuid_pattern[*state][cat];
		return 1;
	} else {
		return 0;
	}
}

/**
 * detect uuids
 */
__global__ void detect(MATCH *odata, const char *idata, int ibuf_sz, unsigned *d_nmatch, UMSTATE *d_umstate, uint16_t nrows) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int iofs = col;
	int oofs = col;
	int stride = gridDim.x * blockDim.x;
	unsigned nmatch = 0;
	uint16_t row;
	UMSTATE &state = d_umstate[col];
	//__syncthreads(); // redundant, as the first thread will always run in an earlier block
	if (col == 0)
		state = d_umstate[STREAMS-1];
	else
		state = 0;
	int lastrow = min(nrows, ibuf_sz-col*nrows);
	for (row=0; row<lastrow; row++,iofs+=stride) {
		if (cuda_um_match(&state, idata[iofs])) {
			odata[oofs] = MATCH {int16_t(row-UMPATLEN+1), UMPATLEN};
			oofs += stride;
			nmatch++;
		}
	}
	if (col < STREAMS-1) {
		iofs = col+1;
		while (state != 0 && row<lastrow+UMPATLEN) {
			if (cuda_um_match(&state, idata[iofs])) {
				odata[oofs] = MATCH {int16_t(row-UMPATLEN+1), UMPATLEN};
				oofs += stride;
				nmatch++;
				break;
			}
			row++;
			iofs+=stride;
		}
	}
	d_nmatch[col] = nmatch;
}

class CudaDetect {
	char *d_ibuf; // original input
	char *d_tibuf; // transposed input
	MATCH *d_tobuf; // transposed output
	MATCH *d_obuf; // regular output
	unsigned *d_nmatch;
	UMSTATE *d_umstate;
public:
	CudaDetect() {
		checkCuda(hipMalloc(&d_ibuf, STREAMS*STRSZ*sizeof(*d_ibuf)));
		checkCuda(hipMalloc(&d_tibuf, STREAMS*STRSZ*sizeof(*d_tibuf)));
		checkCuda(hipMalloc(&d_tobuf, STREAMS*(STRSZ/UMPATLEN+2)*sizeof(*d_tobuf)));
		checkCuda(hipMalloc(&d_obuf, STREAMS*(STRSZ/UMPATLEN+2)*sizeof(*d_tobuf)));
		checkCuda(hipMalloc(&d_nmatch, STREAMS*sizeof(*d_nmatch)));
		checkCuda(hipMalloc(&d_umstate, STREAMS*sizeof(*d_umstate)));
		checkCuda(hipMemset(d_umstate, 0, STREAMS*sizeof(*d_umstate)));
		checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(cuda_uuid_pattern), uuid_pattern, sizeof(uuid_pattern)));
	}
	~CudaDetect() {
		checkCuda(hipFree(d_ibuf));
		checkCuda(hipFree(d_tibuf));
		checkCuda(hipFree(d_tobuf));
		checkCuda(hipFree(d_obuf));
		checkCuda(hipFree(d_nmatch));
		checkCuda(hipFree(d_umstate));
	}
	void operator()(const char *ibuf, int ibuf_sz, MATCH *obuf, unsigned *nmatch, unsigned &rowsz) {
		checkCuda(hipMemcpy(d_ibuf, ibuf, ibuf_sz, hipMemcpyHostToDevice));
		dim3 dimGrid(STRSZ/TRANSPOSE_TILE_DIM, STREAMS/TRANSPOSE_TILE_DIM, 1);
		dim3 dimBlock(TRANSPOSE_TILE_DIM, TRANSPOSE_BLOCK_ROWS, 1);
		transposeNoBankConflicts<<<dimGrid, dimBlock>>>(d_tibuf, d_ibuf);
		checkCuda(hipGetLastError());
		detect<<<STREAMS/THREADS,THREADS>>>(d_tobuf, d_tibuf, ibuf_sz, d_nmatch, d_umstate, STREAMS);
		checkCuda(hipGetLastError());
		checkCuda(hipMemcpy(nmatch, d_nmatch, sizeof(*nmatch)*STREAMS, hipMemcpyDeviceToHost));
		unsigned nmx = rowsz = *std::max_element(nmatch, nmatch+STREAMS);
		if (nmx > 0) {
			rowsz = (nmx+TRANSPOSE_TILE_DIM-1)/TRANSPOSE_TILE_DIM*TRANSPOSE_TILE_DIM;
			dim3 dimGrid(STREAMS/TRANSPOSE_TILE_DIM, rowsz/TRANSPOSE_TILE_DIM, 1);
			dim3 dimBlock(TRANSPOSE_TILE_DIM, TRANSPOSE_BLOCK_ROWS, 1);
			transposeNoBankConflicts<<<dimGrid, dimBlock>>>(d_obuf, d_tobuf);
			checkCuda(hipGetLastError());
			checkCuda(hipMemcpy(obuf, d_obuf, rowsz*STREAMS*sizeof(obuf[0]), hipMemcpyDeviceToHost));
		}
	}
};

class DetectStage: public PipeStageExec {
public:
	struct TRESULT {
		ReadStage::TRESULT in;
		unsigned match_row_sz;
		MATCH *match;
		unsigned *nmatch;
	};
private:
	static constexpr int stages = 2;
	std::unique_ptr<MATCH> up_match[stages];
	std::unique_ptr<unsigned> up_nmatch[stages];
	TRESULT res[stages];
	CudaDetect detect;
	virtual void *next(void *arg) override {
		TRESULT &r = res[batch%stages];
		r.in = *(ReadStage::TRESULT*)arg;
		detect(r.in.buf, r.in.sz, r.match, r.nmatch, r.match_row_sz);
		return &r;
	}
public:
	DetectStage(ReadStage &parent):PipeStageExec(parent),detect() {
		for (int i=0; i<stages; i++) {
			up_match[i] = std::unique_ptr<MATCH>(new MATCH[STRSZ*STREAMS]);
			up_nmatch[i] = std::unique_ptr<unsigned>(new unsigned[STREAMS*stages]);
			res[i] = {ReadStage::TRESULT {nullptr, 0}, 0, up_match[i].get(), up_nmatch[i].get()};
		}
	}
};

void prn(const char *ibuf, const MATCH *match, const unsigned *nmatch, const unsigned match_row_sz) {
	for (int stream=0; stream<STREAMS; stream++) {
		unsigned sz = nmatch[stream];
		const MATCH *mm = match+match_row_sz*stream;
		const char *s = ibuf+STRSZ*stream;
		for (unsigned i=0; i<sz; i++) {
			if ((fwrite(s+mm[i].pos, 1, mm[i].sz, stdout)) != (int)mm[i].sz)
				die("Write error");
			if (fputc('\n', stdout) != '\n')
				die("Write error");
		}
	}
}

int main(int argc, char **argv) {
	ReadStage read(stdin);
	DetectStage detect(read);
	for (auto it:PipeOutput(detect)) {
		DetectStage::TRESULT *r = (DetectStage::TRESULT*)it;
		prn(r->in.buf, r->match, r->nmatch, r->match_row_sz);
	}
}
