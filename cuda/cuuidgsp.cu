#include "hip/hip_runtime.h"
/**
 * similar to uuidgrep, only
 * extracting UUIDs from stream of ASCII strings
 * use STREAMS parallel threads for speed
 * CUDA version of the tool
 * @author Denis Kokarev
 */
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <algorithm>
#include <hip/hip_runtime.h>
#include "transpose.cuh"
#include "uuidmatch.h"
#include "die.h"
#include "par.hh"

constexpr int THREADS = 256;
constexpr int STRSZ = 1<<14; // must be under int16
constexpr int STREAMS = STRSZ;

struct MATCH {
	int16_t pos;
	uint16_t sz;
};

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
#define checkCuda(e) {	if (e!=hipSuccess) { die("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e)); } }

// owns input the data buffers for the pipeline
class ReadStage: public PipeHeadExec {
public:
	struct TRESULT {
		char *buf;
		int sz;
	};
private:
	static constexpr int stages = 4; // need to drag the input over 4 pipe segments
	FILE *fin;
	TRESULT res[stages];
	char *pinned_buf;
private:
	virtual void *next() override {
		fprintf(stderr, "start read batch %d\n", batch);
		if (!feof(fin)) {
			if (batch%stages == 0) // wrap the remaining line around
				memcpy(res[0].buf-UMPATLEN, res[stages-1].buf+STREAMS*STRSZ-UMPATLEN, UMPATLEN);
			TRESULT &r = res[batch%stages];
			r.sz = fread(r.buf, 1, STREAMS*STRSZ, fin);
			if (r.sz < 0)
				die("Read error");
			fprintf(stderr, "end read batch %d\n", batch);
			return &r;
		} else {
			fprintf(stderr, "finished read batch %d\n", batch);
			return nullptr;
		}
	}
public:
	ReadStage(FILE *fin):PipeHeadExec(), fin(fin) {
		checkCuda(hipHostMalloc(&pinned_buf, STRSZ*STREAMS*stages+STRSZ));
		res[0] = {pinned_buf+STRSZ, 0};
		for (int i=1; i<stages; i++)
			res[i] = {res[i-1].buf+STRSZ*STREAMS, 0};
		fprintf(stderr, "initialized read\n");
	}
	~ReadStage() {
		checkCuda(hipHostFree(pinned_buf));
	}
};

inline __device__ int cuda_ch_category(unsigned char c) {
	if ((c>='0' && c<='9') || (c>='a' && c<='f') || (c>='A' && c<='F'))
		return P_HEX;
	else if (c == '-')
		return P_DASH;
	else
		return P_NONE;
}

__constant__ UMSTATE cuda_uuid_pattern[UMPATLEN+1][P_SZ];

// return true when ch was the last char in uuid pattern
inline __device__ int cuda_um_match(UMSTATE *state, char ch) {
	int cat = cuda_ch_category(ch);
	*state = cuda_uuid_pattern[*state][cat];
	if (*state == UMPATLEN) {
		*state = cuda_uuid_pattern[*state][cat];
		return 1;
	} else {
		return 0;
	}
}

/**
 * detect uuids
 */
__global__ void detect(MATCH *odata, const char *idata, int ibuf_sz, unsigned *d_nmatch, UMSTATE *d_umstate, uint16_t nrows) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int iofs = col;
	int oofs = col;
	int stride = gridDim.x * blockDim.x;
	unsigned nmatch = 0;
	uint16_t row;
	UMSTATE &state = d_umstate[col];
	//__syncthreads(); // redundant, as the first thread will always run in an earlier block
	if (col == 0)
		state = d_umstate[STREAMS-1];
	else
		state = 0;
	int lastrow = min(nrows, ibuf_sz-col*nrows);
	for (row=0; row<lastrow; row++,iofs+=stride) {
		if (cuda_um_match(&state, idata[iofs])) {
			odata[oofs] = MATCH {int16_t(row-UMPATLEN+1), UMPATLEN};
			oofs += stride;
			nmatch++;
		}
	}
	if (col < STREAMS-1) {
		iofs = col+1;
		while (state != 0 && row<lastrow+UMPATLEN) {
			if (cuda_um_match(&state, idata[iofs])) {
				odata[oofs] = MATCH {int16_t(row-UMPATLEN+1), UMPATLEN};
				oofs += stride;
				nmatch++;
				break;
			}
			row++;
			iofs+=stride;
		}
	}
	d_nmatch[col] = nmatch;
}

class CudaH2DStage: public PipeStageExec {
public:
	struct TRESULT {
		ReadStage::TRESULT in;
		char *d_ibuf; // device original input
		hipStream_t stream;
	};
private:
	static constexpr int stages = 2;
	TRESULT res[stages];
	virtual void *next(void *arg) override {
		fprintf(stderr, "start h2d batch %d\n", batch);
		TRESULT &r = res[batch%stages];
		r.in = *(ReadStage::TRESULT*)arg;
		checkCuda(hipMemcpyAsync(r.d_ibuf, r.in.buf, r.in.sz, hipMemcpyHostToDevice, r.stream));
		checkCuda(hipStreamSynchronize(r.stream));
		fprintf(stderr, "end h2d batch %d\n", batch);
		return &r;
	}
public:
	CudaH2DStage(PipeHeadExec &parent):PipeStageExec(parent) {
		for (int i=0; i<stages; i++) {
			res[i].in = ReadStage::TRESULT {nullptr, 0};
			checkCuda(hipMalloc(&res[i].d_ibuf, STREAMS*STRSZ*sizeof(*res[i].d_ibuf)));
			checkCuda(hipStreamCreate(&res[i].stream));
		}
		fprintf(stderr, "initialized h2d\n");
	}
	~CudaH2DStage() {
		for (int i=0; i<stages; i++) {
			res[i].in = ReadStage::TRESULT {nullptr, 0};
			checkCuda(hipFree(res[i].d_ibuf));
			checkCuda(hipStreamDestroy(res[i].stream));
		}
	}
};

class CudaDetect {
	char *d_tibuf; // transposed input
	MATCH *d_tobuf; // transposed output
	MATCH *d_obuf; // regular output
	unsigned *d_nmatch;
	UMSTATE *d_umstate;
public:
	CudaDetect() {
		checkCuda(hipMalloc(&d_tibuf, STREAMS*STRSZ*sizeof(*d_tibuf)));
		checkCuda(hipMalloc(&d_tobuf, STREAMS*(STRSZ/UMPATLEN+2)*sizeof(*d_tobuf)));
		checkCuda(hipMalloc(&d_obuf, STREAMS*(STRSZ/UMPATLEN+2)*sizeof(*d_tobuf)));
		checkCuda(hipMalloc(&d_nmatch, STREAMS*sizeof(*d_nmatch)));
		checkCuda(hipMalloc(&d_umstate, STREAMS*sizeof(*d_umstate)));
		checkCuda(hipMemset(d_umstate, 0, STREAMS*sizeof(*d_umstate)));
		checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(cuda_uuid_pattern), uuid_pattern, sizeof(uuid_pattern)));
	}
	~CudaDetect() {
		checkCuda(hipFree(d_tibuf));
		checkCuda(hipFree(d_tobuf));
		checkCuda(hipFree(d_obuf));
		checkCuda(hipFree(d_nmatch));
		checkCuda(hipFree(d_umstate));
		fprintf(stderr, "initialized cuda detect\n");
	}
	void operator()(hipStream_t stream, const char *d_ibuf, int ibuf_sz, MATCH *obuf, unsigned *nmatch, unsigned &rowsz) {
		dim3 dimGrid(STRSZ/TRANSPOSE_TILE_DIM, STREAMS/TRANSPOSE_TILE_DIM, 1);
		dim3 dimBlock(TRANSPOSE_TILE_DIM, TRANSPOSE_BLOCK_ROWS, 1);
		transposeNoBankConflicts<<<dimGrid, dimBlock, 0, stream>>>(d_tibuf, d_ibuf);
		checkCuda(hipGetLastError());
		detect<<<STREAMS/THREADS,THREADS,0,stream>>>(d_tobuf, d_tibuf, ibuf_sz, d_nmatch, d_umstate, STREAMS);
		checkCuda(hipGetLastError());
		checkCuda(hipMemcpyAsync(nmatch, d_nmatch, sizeof(*nmatch)*STREAMS, hipMemcpyDeviceToHost, stream));
		checkCuda(hipStreamSynchronize(stream));
		unsigned nmx = rowsz = *std::max_element(nmatch, nmatch+STREAMS);
		if (nmx > 0) {
			rowsz = (nmx+TRANSPOSE_TILE_DIM-1)/TRANSPOSE_TILE_DIM*TRANSPOSE_TILE_DIM;
			dim3 dimGrid(STREAMS/TRANSPOSE_TILE_DIM, rowsz/TRANSPOSE_TILE_DIM, 1);
			dim3 dimBlock(TRANSPOSE_TILE_DIM, TRANSPOSE_BLOCK_ROWS, 1);
			transposeNoBankConflicts<<<dimGrid, dimBlock, 0, stream>>>(d_obuf, d_tobuf);
			checkCuda(hipGetLastError());
			checkCuda(hipMemcpyAsync(obuf, d_obuf, rowsz*STREAMS*sizeof(obuf[0]), hipMemcpyDeviceToHost, stream));
			checkCuda(hipStreamSynchronize(stream));
		}
	}
};

class DetectStage: public PipeStageExec {
public:
	struct TRESULT {
		CudaH2DStage::TRESULT in;
		unsigned match_row_sz;
		MATCH *match;
		unsigned *nmatch;
	};
private:
	static constexpr int stages = 2;
	TRESULT res[stages];
	CudaDetect detect;
	virtual void *next(void *arg) override {
		fprintf(stderr, "start detect batch %d\n", batch);
		TRESULT &r = res[batch%stages];
		r.in = *(CudaH2DStage::TRESULT*)arg;
		detect(r.in.stream,r.in.d_ibuf, r.in.in.sz, r.match, r.nmatch, r.match_row_sz);
		fprintf(stderr, "end detect batch %d\n", batch);
		return &r;
	}
public:
	DetectStage(PipeHeadExec &parent):PipeStageExec(parent),detect() {
		for (int i=0; i<stages; i++) {
			checkCuda(hipHostMalloc(&res[i].match, sizeof(MATCH)*STRSZ*STREAMS));
			checkCuda(hipHostMalloc(&res[i].nmatch, sizeof(unsigned)*STREAMS));
		}
		fprintf(stderr, "initialized detect\n");
	}
	~DetectStage() {
		for (int i=0; i<stages; i++) {
			checkCuda(hipHostFree(res[i].match));
			checkCuda(hipHostFree(res[i].nmatch));
		}
	}
};

void prn(const char *ibuf, const MATCH *match, const unsigned *nmatch, const unsigned match_row_sz) {
	for (int stream=0; stream<STREAMS; stream++) {
		unsigned sz = nmatch[stream];
		const MATCH *mm = match+match_row_sz*stream;
		const char *s = ibuf+STRSZ*stream;
		for (unsigned i=0; i<sz; i++) {
			if ((fwrite(s+mm[i].pos, 1, mm[i].sz, stdout)) != (int)mm[i].sz)
				die("Write error");
			if (fputc('\n', stdout) != '\n')
				die("Write error");
		}
	}
}

int main(int argc, char **argv) {
	ReadStage read(stdin);
	CudaH2DStage h2d(read);
	DetectStage detect(h2d);
	for (auto it:PipeOutput(detect)) {
		DetectStage::TRESULT *r = (DetectStage::TRESULT*)it;
		prn(r->in.in.buf, r->match, r->nmatch, r->match_row_sz);
	}
}
