#include "hip/hip_runtime.h"
/**
 * similar to uuidgrep, only
 * extracting UUIDs from stream of ASCII strings
 * use STREAMS parallel threads for speed
 * @author Denis Kokarev
 */
#include <cstdio>
#include <cstdlib>
#include <errno.h>
#include "die.h"
#include <memory>
#include <cstring>
#include "uuidmatch.h"
#include <hip/hip_runtime.h>
#include "transpose.cuh"
#include <algorithm>

constexpr int THREADS = 256;
constexpr int STRSZ = 1<<14; // must be under uint16
constexpr int STREAMS = STRSZ;
constexpr char SFILL = ' ';

struct MATCH {
	uint16_t pos;
	uint16_t sz;
};

void prn(const char *ibuf, const MATCH *obuf, const uint16_t *o_sz, const uint16_t rowsz) {
	for (int stream=0; stream<STREAMS; stream++) {
		unsigned sz = o_sz[stream];
		const MATCH *mm = obuf+rowsz*stream;
		const char *s = ibuf+STRSZ*stream;
		for (unsigned i=0; i<sz; i++) {
			if ((fwrite(s+mm[i].pos, 1, mm[i].sz, stdout)) != (int)mm[i].sz)
				die("Write error");
			if (fputc('\n', stdout) != '\n')
				die("Write error");
		}
	}
}

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
static hipError_t checkCuda(hipError_t result) {
	if (result != hipSuccess)
		die("CUDA Runtime Error: %s\n", hipGetErrorString(result));
	return result;
}

// start with automate position 0
inline __device__ void cuda_um_init(UMSTATE *state) {
	*state = 0;
}

inline __device__ int cuda_ch_category(unsigned char c) {
	if ((c>='0' && c<='9') || (c>='a' && c<='f') || (c>='A' && c<='F'))
		return P_HEX;
	else if (c == '-')
		return P_DASH;
	else
		return P_NONE;
}

__constant__ UMSTATE cuda_uuid_pattern[UMPATLEN+1][P_SZ];

// return true when ch was the last char in uuid pattern
inline __device__ int cuda_um_match(UMSTATE *state, char ch) {
	int cat = cuda_ch_category(ch);
	*state = cuda_uuid_pattern[*state][cat];
	if (*state == UMPATLEN) {
		*state = cuda_uuid_pattern[*state][cat];
		return 1;
	} else {
		return 0;
	}
}

/**
 * detect uuids
 */
__global__ void detect(MATCH *odata, const char *idata, uint16_t *d_nmatch, uint16_t nrows) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int iofs = col;
	int oofs = col;
	int stride = gridDim.x * blockDim.x;
	uint16_t nmatch = 0;
	uint16_t row;
	UMSTATE state;
	cuda_um_init(&state);
	for (row=0; row<nrows; row++,iofs+=stride) {
		if (cuda_um_match(&state, idata[iofs])) {
			odata[oofs] = MATCH {uint16_t(row-UMPATLEN+1), UMPATLEN};
			oofs += stride;
			nmatch++;
			//printf("found uuid at row %d\n", row);
		}
	}
	d_nmatch[col] = nmatch;
}

class CudaDetect {
	char *d_ibuf; // original input
	char *d_tibuf; // transposed input
	MATCH *d_tobuf; // transposed output
	MATCH *d_obuf; // regular output
	uint16_t *d_nmatch;
public:
	CudaDetect() {
		checkCuda(hipMalloc(&d_ibuf, STREAMS*STRSZ*sizeof(*d_ibuf)));
		checkCuda(hipMalloc(&d_tibuf, STREAMS*STRSZ*sizeof(*d_tibuf)));
		checkCuda(hipMalloc(&d_tobuf, STREAMS*(STRSZ/UMPATLEN+1)*sizeof(*d_tobuf)));
		checkCuda(hipMalloc(&d_obuf, STREAMS*(STRSZ/UMPATLEN+1)*sizeof(*d_tobuf)));
		checkCuda(hipMalloc(&d_nmatch, STREAMS*sizeof(*d_nmatch)));
		checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(cuda_uuid_pattern), uuid_pattern, sizeof(uuid_pattern)));
	}
	~CudaDetect() {
		checkCuda(hipFree(d_ibuf));
		checkCuda(hipFree(d_tibuf));
		checkCuda(hipFree(d_tobuf));
		checkCuda(hipFree(d_obuf));
		checkCuda(hipFree(d_nmatch));
	}
	void operator()(const char *ibuf, MATCH *obuf, uint16_t *nmatch, uint16_t &rowsz) {
		checkCuda(hipMemcpy(d_ibuf, ibuf, sizeof(*ibuf)*STRSZ*STREAMS, hipMemcpyHostToDevice));
		dim3 dimGrid(STRSZ/TRANSPOSE_TILE_DIM, STREAMS/TRANSPOSE_TILE_DIM, 1);
		dim3 dimBlock(TRANSPOSE_TILE_DIM, TRANSPOSE_BLOCK_ROWS, 1);
		transposeNoBankConflicts<<<dimGrid, dimBlock>>>(d_tibuf, d_ibuf);
		checkCuda(hipGetLastError());
		detect<<<STREAMS/THREADS,THREADS>>>(d_tobuf, d_tibuf, d_nmatch, STREAMS);
		checkCuda(hipGetLastError());
		checkCuda(hipMemcpy(nmatch, d_nmatch, sizeof(*nmatch)*STREAMS, hipMemcpyDeviceToHost));
#if 0
		for (int i=0; i<STREAMS; i++)
			fprintf(stderr, "nmatch[%d] = %d\n", i, nmatch[i]);
#endif
		uint16_t nmx = rowsz = *std::max_element(nmatch, nmatch+STREAMS);
		if (nmx > 0) {
			rowsz = (nmx+TRANSPOSE_TILE_DIM-1)/TRANSPOSE_TILE_DIM*TRANSPOSE_TILE_DIM;
			dim3 dimGrid(STREAMS/TRANSPOSE_TILE_DIM, rowsz/TRANSPOSE_TILE_DIM, 1);
			dim3 dimBlock(TRANSPOSE_TILE_DIM, TRANSPOSE_BLOCK_ROWS, 1);
			transposeNoBankConflicts<<<dimGrid, dimBlock>>>(d_obuf, d_tobuf);
			checkCuda(hipGetLastError());
			checkCuda(hipMemcpy(obuf, d_obuf, rowsz*STREAMS*sizeof(obuf[0]), hipMemcpyDeviceToHost));
		}
	}
};

inline int rfindnl(const char *buf, int sz) {
	int over = 0;
	while (sz > 0 && buf[sz-1] != '\n')
		sz--, over++;
	return over;
}

int main(int argc, char **argv) {
	int over = 0;
	std::unique_ptr<char[]> up_ibuf(new char[STRSZ*(STREAMS+1)]); // +1 to carry over the remaining of the line
	std::unique_ptr<MATCH[]> up_obuf(new MATCH[STRSZ*(STREAMS)]);
	char *ibuf = up_ibuf.get();
	MATCH *obuf = up_obuf.get();
	uint16_t nmatch[STREAMS];
	CudaDetect cuda_detect;
	// read input and scatter it into STREAMS channels
	while (!feof(stdin)) {
		int ns = 0;
		char *buf = ibuf;
		memcpy(buf, buf+STREAMS*STRSZ, over);
		while (ns < STREAMS) {
			int rsz = STRSZ-over;
			int rc = fread(buf+over, 1, rsz, stdin);
			if (rc == rsz) {
				over = rfindnl(buf, STRSZ);
				if (over == STRSZ)
					die("Line size must be less than %d", STRSZ);
				char *next_buf = buf+STRSZ;
				memcpy(next_buf, buf+STRSZ-over, over);
				memset(buf+STRSZ-over, SFILL, over);
				buf = next_buf;
				ns++;
			} else if (rc < 0) {
				die("read error");
			} else {
				// eof - we didn't get full block
				memset(buf+over+rc, SFILL, STRSZ-over-rc);
				ns++;
				memset(ibuf+STRSZ*ns, SFILL, (STREAMS-ns)*STRSZ); // fill up all unused channels
				break;
			}
		}
		//////// run the batch
		uint16_t rowsz;
		cuda_detect(ibuf, obuf, nmatch, rowsz);
		prn(ibuf, obuf, nmatch, rowsz);
	}
}
