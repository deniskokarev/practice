#include "hip/hip_runtime.h"
/**
 * similar to uuidgrep, only
 * extracting UUIDs from stream of ASCII strings
 * use STREAMS parallel threads for speed
 * CUDA version of the tool
 * @author Denis Kokarev
 */
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <memory>
#include <algorithm>
#include <hip/hip_runtime.h>
#include "transpose.cuh"
#include "uuidmatch.h"
#include "die.h"

constexpr int THREADS = 256;
constexpr int STRSZ = 1<<14; // must be under int16
constexpr int STREAMS = STRSZ;

struct MATCH {
	int16_t pos;
	uint16_t sz;
};

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
static hipError_t checkCuda(hipError_t result) {
	if (result != hipSuccess)
		die("CUDA Runtime Error: %s\n", hipGetErrorString(result));
	return result;
}

inline __device__ int cuda_ch_category(unsigned char c) {
	if ((c>='0' && c<='9') || (c>='a' && c<='f') || (c>='A' && c<='F'))
		return P_HEX;
	else if (c == '-')
		return P_DASH;
	else
		return P_NONE;
}

__constant__ UMSTATE cuda_uuid_pattern[UMPATLEN+1][P_SZ];

// return true when ch was the last char in uuid pattern
inline __device__ int cuda_um_match(UMSTATE *state, char ch) {
	int cat = cuda_ch_category(ch);
	*state = cuda_uuid_pattern[*state][cat];
	if (*state == UMPATLEN) {
		*state = cuda_uuid_pattern[*state][cat];
		return 1;
	} else {
		return 0;
	}
}

/**
 * detect uuids
 */
__global__ void detect(MATCH *odata, const char *idata, int ibuf_sz, uint16_t *d_nmatch, UMSTATE *d_umstate, uint16_t nrows) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int iofs = col;
	int oofs = col;
	int stride = gridDim.x * blockDim.x;
	uint16_t nmatch = 0;
	uint16_t row;
	UMSTATE &state = d_umstate[col];
	//__syncthreads(); // redundant, as the first thread will always run in an earlier block
	if (col == 0)
		state = d_umstate[STREAMS-1];
	else
		state = 0;
	int lastrow = min(nrows, ibuf_sz-col*nrows);
	for (row=0; row<lastrow; row++,iofs+=stride) {
		if (cuda_um_match(&state, idata[iofs])) {
			odata[oofs] = MATCH {int16_t(row-UMPATLEN+1), UMPATLEN};
			oofs += stride;
			nmatch++;
		}
	}
	if (col < STREAMS-1) {
		iofs = col+1;
		while (state != 0 && row<lastrow+UMPATLEN) {
			if (cuda_um_match(&state, idata[iofs])) {
				odata[oofs] = MATCH {int16_t(row-UMPATLEN+1), UMPATLEN};
				oofs += stride;
				nmatch++;
				break;
			}
			row++;
			iofs+=stride;
		}
	}
	d_nmatch[col] = nmatch;
}

class CudaDetect {
	char *d_ibuf; // original input
	char *d_tibuf; // transposed input
	MATCH *d_tobuf; // transposed output
	MATCH *d_obuf; // regular output
	uint16_t *d_nmatch;
	UMSTATE *d_umstate;
public:
	CudaDetect() {
		checkCuda(hipMalloc(&d_ibuf, STREAMS*STRSZ*sizeof(*d_ibuf)));
		checkCuda(hipMalloc(&d_tibuf, STREAMS*STRSZ*sizeof(*d_tibuf)));
		checkCuda(hipMalloc(&d_tobuf, STREAMS*(STRSZ/UMPATLEN+2)*sizeof(*d_tobuf)));
		checkCuda(hipMalloc(&d_obuf, STREAMS*(STRSZ/UMPATLEN+2)*sizeof(*d_tobuf)));
		checkCuda(hipMalloc(&d_nmatch, STREAMS*sizeof(*d_nmatch)));
		checkCuda(hipMalloc(&d_umstate, STREAMS*sizeof(*d_umstate)));
		checkCuda(hipMemset(d_umstate, 0, STREAMS*sizeof(*d_umstate)));
		checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(cuda_uuid_pattern), uuid_pattern, sizeof(uuid_pattern)));
	}
	~CudaDetect() {
		checkCuda(hipFree(d_ibuf));
		checkCuda(hipFree(d_tibuf));
		checkCuda(hipFree(d_tobuf));
		checkCuda(hipFree(d_obuf));
		checkCuda(hipFree(d_nmatch));
		checkCuda(hipFree(d_umstate));
	}
	void operator()(const char *ibuf, int ibuf_sz, MATCH *obuf, uint16_t *nmatch, uint16_t &rowsz) {
		checkCuda(hipMemcpy(d_ibuf, ibuf, ibuf_sz, hipMemcpyHostToDevice));
		dim3 dimGrid(STRSZ/TRANSPOSE_TILE_DIM, STREAMS/TRANSPOSE_TILE_DIM, 1);
		dim3 dimBlock(TRANSPOSE_TILE_DIM, TRANSPOSE_BLOCK_ROWS, 1);
		transposeNoBankConflicts<<<dimGrid, dimBlock>>>(d_tibuf, d_ibuf);
		checkCuda(hipGetLastError());
		detect<<<STREAMS/THREADS,THREADS>>>(d_tobuf, d_tibuf, ibuf_sz, d_nmatch, d_umstate, STREAMS);
		checkCuda(hipGetLastError());
		checkCuda(hipMemcpy(nmatch, d_nmatch, sizeof(*nmatch)*STREAMS, hipMemcpyDeviceToHost));
		uint16_t nmx = rowsz = *std::max_element(nmatch, nmatch+STREAMS);
		if (nmx > 0) {
			rowsz = (nmx+TRANSPOSE_TILE_DIM-1)/TRANSPOSE_TILE_DIM*TRANSPOSE_TILE_DIM;
			dim3 dimGrid(STREAMS/TRANSPOSE_TILE_DIM, rowsz/TRANSPOSE_TILE_DIM, 1);
			dim3 dimBlock(TRANSPOSE_TILE_DIM, TRANSPOSE_BLOCK_ROWS, 1);
			transposeNoBankConflicts<<<dimGrid, dimBlock>>>(d_obuf, d_tobuf);
			checkCuda(hipGetLastError());
			checkCuda(hipMemcpy(obuf, d_obuf, rowsz*STREAMS*sizeof(obuf[0]), hipMemcpyDeviceToHost));
		}
	}
};

void prn(const char *ibuf, const MATCH *obuf, const uint16_t *o_sz, const uint16_t rowsz) {
	for (int stream=0; stream<STREAMS; stream++) {
		unsigned sz = o_sz[stream];
		const MATCH *mm = obuf+rowsz*stream;
		const char *s = ibuf+STRSZ*stream;
		for (unsigned i=0; i<sz; i++) {
			if ((fwrite(s+mm[i].pos, 1, mm[i].sz, stdout)) != (int)mm[i].sz)
				die("Write error");
			if (fputc('\n', stdout) != '\n')
				die("Write error");
		}
	}
}

inline int rfindnl(const char *buf, int sz) {
	int over = 0;
	while (sz > 0 && buf[sz-1] != '\n')
		sz--, over++;
	return over;
}

int main(int argc, char **argv) {
	std::unique_ptr<char[]> up_ibuf(new char[STRSZ*STREAMS+STRSZ]); // +1 to carry over the remaining of the line
	std::unique_ptr<MATCH[]> up_obuf(new MATCH[STRSZ*STREAMS]);
	char *ibuf = up_ibuf.get()+STRSZ;
	MATCH *obuf = up_obuf.get();
	uint16_t nmatch[STREAMS];
	CudaDetect detect;
	// read raw input
	while (!feof(stdin)) {
		memcpy(ibuf-UMPATLEN, ibuf+STREAMS*STRSZ-UMPATLEN, UMPATLEN);
		int sz = fread(ibuf, 1, STREAMS*STRSZ, stdin);
		if (sz < 0)
			die("read error");
		//////// run the batch
		uint16_t rowsz;
		detect(ibuf, sz, obuf, nmatch, rowsz);
		prn(ibuf, obuf, nmatch, rowsz);
	}
}
