#include "hip/hip_runtime.h"
/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_runtime.h>
#include "transpose.hh"
#include "die.h"

// 256 threads altogether
const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
static hipError_t checkCuda(hipError_t result) {
	if (result != hipSuccess)
		die("CUDA Runtime Error: %s\n", hipGetErrorString(result));
	return result;
}

__global__ void copy(char *odata, const char *idata)
{
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
    odata[(y+j)*width + x] = idata[(y+j)*width + x];
}

// No bank-conflict transpose
// Same as transposeCoalesced except the first tile dimension is padded 
// to avoid shared memory bank conflicts.
__global__ void transposeNoBankConflicts(char *odata, const char *idata)
{
  __shared__ char tile[TILE_DIM][TILE_DIM+1];
    
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}

Match::CudaTranspose::CudaTranspose(const char *ib, char *ob, int d):d_obuf(nullptr),d_ibuf(nullptr),ibuf(ib),obuf(ob),dim(d),err(nullptr) {
	if (d % TILE_DIM == 0) {
		hipError_t rc;
		if ((rc=hipMalloc(&d_ibuf, dim*dim))!=0)
			err = hipGetErrorString(rc);
		if ((rc=hipMalloc(&d_obuf, dim*dim))!=0)
			err = hipGetErrorString(rc);
	} else {
		err = "Dimention must be divisible by TILE_DIM";
	}
}

Match::CudaTranspose::operator bool() const {
	return err == nullptr;
}

Match::CudaTranspose::~CudaTranspose() {
	if (d_obuf)
		checkCuda(hipFree((void*)d_obuf));
	if (d_ibuf)
		checkCuda(hipFree((void*)d_ibuf));
}

void Match::CudaTranspose::run() {
	dim3 dimGrid(dim/TILE_DIM, dim/TILE_DIM, 1);
	dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);
	checkCuda(hipMemcpy(d_ibuf, ibuf, dim*dim, hipMemcpyHostToDevice));
	transposeNoBankConflicts<<<dimGrid, dimBlock>>>((char*)d_obuf, (const char*)d_ibuf);
	checkCuda(hipGetLastError());
	//checkCuda(hipMemcpy(obuf, d_obuf, dim*dim, hipMemcpyDeviceToHost));
}
