#include "hip/hip_runtime.h"
/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_runtime.h>
#include "transpose.hh"
#include "die.h"

// 256 threads altogether
const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
static hipError_t checkCuda(hipError_t result) {
	if (result != hipSuccess)
		die("CUDA Runtime Error: %s\n", hipGetErrorString(result));
	return result;
}

// No bank-conflict transpose
// Same as transposeCoalesced except the first tile dimension is padded 
// to avoid shared memory bank conflicts.
__global__ void transposeNoBankConflicts(char *odata, const char *idata)
{
  __shared__ char tile[TILE_DIM][TILE_DIM+1];
    
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}

Match::CudaTranspose::CudaTranspose() {
	was_init = false;
}

int Match::CudaTranspose::init(const char *ib, char *ob, int d) {
	if (d % TILE_DIM != 0)
		return -1;
	ibuf = ib;
	obuf= ob;
	dim = d;
	int rc;
	if ((rc=hipMalloc(&d_ibuf, dim*dim))!=0)
		return rc;
	if ((rc=hipMalloc(&d_obuf, dim*dim))!=0)
		return rc;
	was_init = true;
	return 0;
}

Match::CudaTranspose::~CudaTranspose() {
	if (was_init) {
		checkCuda(hipFree((void*)d_ibuf));
		checkCuda(hipFree((void*)d_obuf));
	}
}

void Match::CudaTranspose::run() {
	dim3 dimGrid(dim/TILE_DIM, dim/TILE_DIM, 1);
	dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);
	checkCuda(hipMemcpy(d_ibuf, ibuf, dim*dim, hipMemcpyHostToDevice));
	transposeNoBankConflicts<<<dimGrid, dimBlock>>>(d_obuf, d_ibuf);
	checkCuda(hipMemcpy(obuf, d_obuf, dim*dim, hipMemcpyDeviceToHost));
}
